#include "hip/hip_runtime.h"
#include "naive_cuda_simulation.cuh"
#include "physics/gravitation.h"
#include "physics/mechanics.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cuda_wrappers.cuh"
#include "constants.h"

std::vector<double2> vector_map_vector2d_to_double2(const std::vector<Vector2d<double>> vector) {
    std::vector<double2> returnVector (vector.size());

    for (size_t i = 0; i < vector.size(); ++i) {
        returnVector[i] = double2{vector[i][0], vector[i][1]};
    }

    return returnVector;
}

void NaiveCudaSimulation::allocate_device_memory(Universe& universe, void** d_weights, void** d_forces, void** d_velocities, void** d_positions){

    size_t number_bodies = universe.num_bodies;
    size_t memory_size_weights = number_bodies * sizeof(double);
    size_t memory_size_vectors = number_bodies * sizeof(double2);

    // alloc weight memory 
    parprog_cudaMalloc(d_weights, memory_size_weights);

    // alloc vector memory
    parprog_cudaMalloc(d_forces, memory_size_vectors);
    parprog_cudaMalloc(d_velocities, memory_size_vectors);
    parprog_cudaMalloc(d_positions, memory_size_vectors);
}

void NaiveCudaSimulation::free_device_memory(void** d_weights, void** d_forces, void** d_velocities, void** d_positions){

    // free memory 
    parprog_cudaFree(*d_weights);
    parprog_cudaFree(*d_forces);
    parprog_cudaFree(*d_velocities);
    parprog_cudaFree(*d_positions);

    // avoid dangling pointers 
    d_weights = nullptr;
    d_forces = nullptr;
    d_velocities = nullptr;
    d_positions = nullptr;
}

void NaiveCudaSimulation::copy_data_to_device(Universe& universe, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){
    parprog_cudaMemcpy(d_weights, universe.weights.data(), universe.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    parprog_cudaMemcpy(d_forces, vector_map_vector2d_to_double2(universe.forces).data(), universe.num_bodies * sizeof(double2), hipMemcpyHostToDevice);
    parprog_cudaMemcpy(d_velocities, vector_map_vector2d_to_double2(universe.velocities).data(), universe.num_bodies * sizeof(double2), hipMemcpyHostToDevice);
    parprog_cudaMemcpy(d_positions, vector_map_vector2d_to_double2(universe.positions).data(), universe.num_bodies * sizeof(double2), hipMemcpyHostToDevice);
}

void NaiveCudaSimulation::copy_data_from_device(Universe& universe, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){
    parprog_cudaMemcpy(universe.weights.data(), d_weights, universe.num_bodies * sizeof(double), hipMemcpyDeviceToHost);

    double2* forces = (double2*) malloc(universe.num_bodies * sizeof(double2));
    parprog_cudaMemcpy(forces, d_forces, universe.num_bodies * sizeof(double2), hipMemcpyDeviceToHost);
    for (int i = 0; i < universe.num_bodies; ++i) {
        double2 cur = forces[i];
        universe.forces[i] = Vector2d<double>(cur.x, cur.y);
    }
    free(forces);

    double2* velocities = (double2*) malloc(universe.num_bodies * sizeof(double2));
    parprog_cudaMemcpy(velocities, d_velocities, universe.num_bodies * sizeof(double2), hipMemcpyDeviceToHost);
    for (int i = 0; i < universe.num_bodies; ++i) {
        double2 cur = velocities[i];
        universe.velocities[i] = Vector2d<double>(cur.x, cur.y);
    }
    free(velocities);

    double2* positions = (double2*) malloc(universe.num_bodies * sizeof(double2));
    parprog_cudaMemcpy(positions, d_positions, universe.num_bodies * sizeof(double2), hipMemcpyDeviceToHost);
    for (int i = 0; i < universe.num_bodies; ++i) {
        double2 cur = positions[i];
        universe.positions[i] = Vector2d<double>(cur.x, cur.y);
    }
    free(positions);
}

__global__
void calculate_forces_kernel(std::uint32_t num_bodies, double2* d_positions, double* d_weights, double2* d_forces){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_bodies) return;

    double2 i_position = d_positions[i];
    double i_weight = d_weights[i];

    double2 total_force {0, 0};
    for (int j = 0; j < num_bodies; ++j) {
        if (j == i) continue;

        double2 j_position = d_positions[j];
        double j_weight = d_weights[j];
        double2 direction {j_position.x - i_position.x, j_position.y - i_position.y};

        double distance = sqrt(pow(direction.x, 2) + pow(direction.y, 2));
        double force = gravitational_constant * ((i_weight * j_weight)/(pow(distance, 2)));
        double unit_vector_force = force / distance;

        total_force.x += direction.x * unit_vector_force;
        total_force.y += direction.y * unit_vector_force;
    }

    d_forces[i] = total_force;
}

void NaiveCudaSimulation::calculate_forces(Universe& universe, void* d_positions, void* d_weights, void* d_forces){
    int block_size = 512;
    int grid_size = universe.num_bodies % block_size == 0 ? universe.num_bodies / block_size : (universe.num_bodies - (universe.num_bodies % block_size) + block_size) / block_size;
    
    dim3 block_dim(block_size);
    dim3 grid_dim(grid_size);
    calculate_forces_kernel<<<grid_dim, block_dim>>>(universe.num_bodies, (double2*) d_positions, (double*) d_weights, (double2*) d_forces);;
}

__global__
void calculate_velocities_kernel(std::uint32_t num_bodies, double2* d_forces, double* d_weights, double2* d_velocities){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_bodies) return;

    double m = d_weights[i];
    double2 force = d_forces[i];
    double2 v0 = d_velocities[i];
    double2 a = {force.x / m, force.y / m};
    d_velocities[i] = {v0.x + a.x * epoch_in_seconds, v0.y + a.y * epoch_in_seconds};
}

void NaiveCudaSimulation::calculate_velocities(Universe& universe, void* d_forces, void* d_weights, void* d_velocities){
    int block_size = 512;
    int grid_size = universe.num_bodies % block_size == 0 ? universe.num_bodies / block_size : (universe.num_bodies - (universe.num_bodies % block_size) + block_size) / block_size;

    dim3 block_dim(block_size);
    dim3 grid_dim(grid_size);

    calculate_velocities_kernel<<<grid_dim, block_dim>>>(universe.num_bodies, (double2 *)d_forces, (double *)d_weights, (double2 *)d_velocities);
}

__global__
void calculate_positions_kernel(std::uint32_t num_bodies, double2* d_velocities, double2* d_positions){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_bodies) return;

    double2 pos = d_positions[i];
    double2 vel = d_velocities[i];
    d_positions[i] = {pos.x + vel.x * epoch_in_seconds, pos.y + vel.y * epoch_in_seconds};
}

void NaiveCudaSimulation::calculate_positions(Universe& universe, void* d_velocities, void* d_positions){
    int block_size = 512;
    int grid_size = universe.num_bodies % block_size == 0 ? universe.num_bodies / block_size : (universe.num_bodies - (universe.num_bodies % block_size) + block_size) / block_size;

    dim3 block_dim(block_size);
    dim3 grid_dim(grid_size);
    calculate_positions_kernel<<<grid_dim, block_dim>>>(universe.num_bodies, (double2 *)d_velocities, (double2 *)d_positions);
}

void NaiveCudaSimulation::simulate_epochs(Plotter& plotter, Universe& universe, std::uint32_t num_epochs, bool create_intermediate_plots, std::uint32_t plot_intermediate_epochs){

    void* d_weights;
    void* d_forces;
    void* d_velocities;
    void* d_positions;

    allocate_device_memory(universe, &d_weights, &d_forces, &d_velocities, &d_positions);
    
    for (int i = 0; i < num_epochs; i++)
    {
        simulate_epoch(plotter, universe, create_intermediate_plots, plot_intermediate_epochs, d_weights, d_forces, d_velocities, d_positions);
    }

    free_device_memory(d_weights, d_forces, d_velocities, d_positions);
}

__global__
void get_pixels_kernel(std::uint32_t num_bodies, double2* d_positions, std::uint8_t* d_pixels, std::uint32_t plot_width, std::uint32_t plot_height, double plot_bounding_box_x_min, double plot_bounding_box_x_max, double plot_bounding_box_y_min, double plot_bounding_box_y_max){
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // check if within bounding box
    if (d_positions[idx].x >= plot_bounding_box_x_min && d_positions[idx].x <= plot_bounding_box_x_max && d_positions[idx].y >= plot_bounding_box_y_min && d_positions[idx].y <= plot_bounding_box_y_max)
    {
        int pixel_x = (d_positions[idx].x - plot_bounding_box_x_min) / (plot_bounding_box_x_max - plot_bounding_box_x_min) * (plot_width - 1);
        int pixel_y = (d_positions[idx].y - plot_bounding_box_y_min) / (plot_bounding_box_y_max - plot_bounding_box_y_min) * (plot_height - 1);

        // write to 255 to corresponding pixel

        d_pixels[pixel_y * plot_width + pixel_x] = 255;
    }
    // just keep the zero
}

std::vector<std::uint8_t> NaiveCudaSimulation::get_pixels(std::uint32_t plot_width, std::uint32_t plot_height, BoundingBox plot_bounding_box, void* d_positions, std::uint32_t num_bodies){
    // allocate memory
    void* d_pixels;
    uint32_t number_pixels = plot_width * plot_height;
    parprog_cudaMalloc(&d_pixels, number_pixels * sizeof(uint8_t));

    std::vector<std::uint8_t> pixels;
    pixels.resize(number_pixels, 0);

    // call get_pixels_kernel (write either 1 or zero)

    dim3 blockDim(num_bodies, 1, 1);
    dim3 gridDim(1, 1);
    get_pixels_kernel<<<gridDim, blockDim>>>(num_bodies, reinterpret_cast<double2*>(d_positions), reinterpret_cast<uint8_t*>(d_pixels),
        plot_width, plot_height, plot_bounding_box.x_min, plot_bounding_box.x_max, plot_bounding_box.y_min, plot_bounding_box.y_max);

    // copy back from device
    parprog_cudaMemcpy(pixels.data(), &d_pixels, plot_width * plot_height * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // free memory
    parprog_cudaFree(d_pixels);

    return pixels;
}

__global__
void compress_pixels_kernel(std::uint32_t num_raw_pixels, std::uint8_t* d_raw_pixels, std::uint8_t* d_compressed_pixels){
    // basically reduction algorithm
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // for every compressed pixel
    uint8_t sum = 0;
    // iterate over 8 pixels
    for (uint8_t i = 0; i < 8; i++)
    {
        if (idx * 8 + i >= num_raw_pixels)
            break;
        if (d_raw_pixels[idx * 8 + i] != 0)
        {
            sum += (1 << i);
        }
    }

    d_compressed_pixels[idx] = sum;
}

void NaiveCudaSimulation::compress_pixels(std::vector<std::uint8_t>& raw_pixels, std::vector<std::uint8_t>& compressed_pixels){
    
    // allocate memory
    void* d_raw_pixels;
    void* d_compressed_pixels;

    size_t number_raw_pixels = raw_pixels.size();
    size_t number_comp_pixels = compressed_pixels.size();

    int block_size = 512;
    int grid_size = number_compressed_pixels % block_size == 0 ? number_compressed_pixels / block_size : (number_compressed_pixels - (number_compressed_pixels % block_size) + block_size) / block_size;

    parprog_cudaMalloc(&d_raw_pixels, number_raw_pixels * sizeof(uint8_t));
    parprog_cudaMalloc(&d_compressed_pixels, number_comp_pixels * sizeof(uint8_t));

    // copy to device
    parprog_cudaMemcpy(d_raw_pixels, raw_pixels.data(), number_raw_pixels * sizeof(uint8_t), hipMemcpyHostToDevice);
    // parprog_cudaMemcpy(d_compressed_pixels, &raw_pixels, number_comp_pixels * sizeof(uint8_t), hipMemcpyHostToDevice);

    dim3 blockDim(block_size);
    dim3 gridDim(grid_size);

    // call kernel
    compress_pixels_kernel<<<gridDim, blockDim>>>(number_raw_pixels, reinterpret_cast<uint8_t*>(d_raw_pixels), reinterpret_cast<uint8_t*>(d_compressed_pixels));

    // copy back to host
    parprog_cudaMemcpy(compressed_pixels.data(), d_compressed_pixels, number_comp_pixels * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // free memory
    parprog_cudaFree(d_raw_pixels);
    parprog_cudaFree(d_compressed_pixels);
}

void NaiveCudaSimulation::simulate_epoch(Plotter& plotter, Universe& universe, bool create_intermediate_plots, std::uint32_t plot_intermediate_epochs, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){
    calculate_forces(universe, d_positions, d_weights, d_forces);
    calculate_velocities(universe, d_forces, d_weights, d_velocities);
    calculate_positions(universe, d_velocities, d_positions);

    universe.current_simulation_epoch++;
    if(create_intermediate_plots){
        if(universe.current_simulation_epoch % plot_intermediate_epochs == 0){
            std::vector<std::uint8_t> pixels = get_pixels(plotter.get_plot_width(), plotter.get_plot_height(), plotter.get_plot_bounding_box(), d_positions, universe.num_bodies);
            plotter.add_active_pixels_to_image(pixels);

            // This is a dummy to use compression in plotting, although not beneficial performance-wise
            // ----
            // std::vector<std::uint8_t> compressed_pixels;
            // compressed_pixels.resize(pixels.size()/8);
            // compress_pixels(pixels, compressed_pixels);
            // plotter.add_compressed_pixels_to_image(compressed_pixels);
            // ----

            plotter.write_and_clear();
        }
    }
}

void NaiveCudaSimulation::calculate_forces_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_positions, void* d_weights, void* d_forces){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_forces_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_positions, (double*) d_weights, (double2*) d_forces);
}

void NaiveCudaSimulation::calculate_velocities_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_forces, void* d_weights, void* d_velocities){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_velocities_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_forces, (double*) d_weights, (double2*) d_velocities);
}

void NaiveCudaSimulation::calculate_positions_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_velocities, void* d_positions){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_positions_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_velocities, (double2*) d_positions);
}
